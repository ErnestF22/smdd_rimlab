#include "hip/hip_runtime.h"
#include <boost/lexical_cast.hpp>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <string>

#include <pcl/io/pcd_io.h>
#include <pcl/point_types.h>
#include <pcl/visualization/pcl_visualizer.h>

#include <gme_GaussianMixtureEstimator.h>

#include <pcl/features/normal_3d.h>
#include <pcl/io/pcd_io.h>
#include <pcl/point_types.h>
#include <pcl/visualization/pcl_visualizer.h>

#include <rofl/common/macros.h>
#include <rofl/common/param_map.h>
#include <rofl/common/profiler.h>

// #include <gme_GaussianMixtureEstimator.h>
#include <bin_utils.h>
#include <dsd_utils.h>
#include <gme_gaussian_metric.h>

// CUDA
#include <dsd_utils.cuh>
#include <>
#include <thrust/host_vector.h>

int main(int argc, char **argv)
{
    std::string filenameCfg, filePath;
    std::string filenameIn, filenameOutIse, filenameOutRtc;
    std::set<fs::path> sortedByName;

    double sigmaIn;
    std::vector<double> sigmasIn;
    std::vector<double> weightsIn;
    bool enableGmmHier;
    bool plotEllipsesArrows;

    double sigmaMin, res, iseThresh;

    double angleWin;
    int nSamples;
    bool enablePclVisualization;

    int numThreads;

    rofl::ParamMap params;

    params.read(argc, argv);
    params.getParam<std::string>("cfg", filenameCfg, std::string(""));
    params.read(filenameCfg);
    params.read(argc, argv);
    // Output mode (quat or aa)
    params.getParam<std::string>(
        "in", filenameIn,
        std::string("/Geode/Urban_tunnel_01/LiDAR/bin/"));
    params.getParam<std::string>("out_ise", filenameOutIse,
                                 std::string("ise.csv"));
    params.getParam<std::string>("out_rtc", filenameOutRtc,
                                 std::string("rtc.csv"));
    params.getParam<double>("sigmaIn", sigmaIn, 0.10);
    params.getParam<double>("sigmaMin", sigmaMin, 0.05);
    params.getParam<double>("res", res, 1);
    params.getParam<double>("iseThresh", iseThresh, 0.2);
    params.getParam<double>("angleWin", angleWin, M_PI / 180.0 * 5.0);
    params.getParam<int>("nSamples", nSamples, 180);
    params.getParam<bool>("enableGmmHier", enableGmmHier, true);
    params.getParam<bool>("plotEllipsesArrows", plotEllipsesArrows, false);
    params.getParam<int>("numThreads", numThreads, 256);
    params.getParam<bool>("enablePclVisualization", enablePclVisualization, false);

    params.adaptTildeInPaths();
    params.getParam<std::string>("in", filenameIn, "sample.csv");

    std::cout << "Params:" << std::endl;
    params.write(std::cout);

    std::cout << "-------\n"
              << std::endl;

    BinReader binReader;
    binReader.setVehiclePtsMinMax(dsd::Vector2(-3.0, 3.0),
                                  dsd::Vector2(-3.0, 3.0),
                                  dsd::Vector2(-3.0, 3.0));
    binReader.readCloudBin(filenameIn, BinReader::LidarType::VELODYNE);

    dsd::VectorVector3 musIn = binReader.getCloud();

    pcl::PointCloud<pcl::PointXYZ>::Ptr cloud(
        new pcl::PointCloud<pcl::PointXYZ>);
    dsd::binToPcl(musIn, cloud); // just for visualization purposes

    std::cout << "Read " << musIn.size() << " points" << std::endl;

    size_t n = musIn.size();
    for (size_t i = 0; i < n; ++i)
    {
        // ROFL_VAR2(i, musIn[i].transpose())
    }

    sigmasIn.resize(n);
    std::fill(sigmasIn.begin(), sigmasIn.end(), sigmaIn * sigmaIn);
    weightsIn.resize(n);
    std::fill(weightsIn.begin(), weightsIn.end(), 1.0 / n);

    ////////////////////////////////////////////////////////////////////////

    double execTimeI = 0.0;

    dsd::VectorVector3 musOut;
    dsd::VectorMatrix3 sigmasOut;
    std::vector<double> weightsOut;

    /* Calling GMM Hierarchical Estimation */
    if (enableGmmHier)
    {
        rofl::ScopedTimer gmeTimer("gme hier timer");

        gme::GaussianMixtureEstimatorHierarchical3d gme;
        gme.setSigmaMin(sigmaMin);
        // gme.setCovarWidth(sigmaMin); // not used
        gme.setIseThreshold(iseThresh);
        gme.setCellSizeMax(res);
        gme.compute(musIn);

        // gme.initIsotropic(musIn);
        gme.exportGaussians(musOut, sigmasOut, weightsOut);
        ROFL_VAR3(musOut.size(), sigmasOut.size(), weightsOut.size());
        execTimeI += gmeTimer.elapsedTimeMs();
        ROFL_VAR1(gmeTimer.elapsedTimeMs())
    }
    rofl::Profiler::getProfiler().printStats(std::cout);

    // std::cout << "Loading point cloud from \"" << filenameIn << "\""
    //           << std::endl;
    // if (pcl::io::loadPCDFile(filenameIn, *cloud) < 0)
    // {
    //     std::cerr << "Cannot load point cloud from \"" << filenameIn
    //               << "\"" << std::endl;
    //     return 1;
    // }

    size_t cloudSz = cloud->size();
    std::cout << "PCL cloud has " << cloudSz << " points" << std::endl;

    std::for_each(weightsOut.begin(), weightsOut.end(),
                  [cloudSz](double &n)
                  { n *= cloudSz; });

    //
    dsd::VectorVector3 musVmm;
    std::vector<double> kappasVmm;
    std::vector<double> weightsVmm;
    int szPadded = dsd::computeSzPadded(musOut.size(), nSamples, numThreads);
    ROFL_VAR1("von Mises stats")
    {
        rofl::ScopedTimer vonMisesStats("von Mises stats");

        dsd::vonMisesStats3dCuda(musVmm, kappasVmm, weightsVmm, szPadded, musOut, sigmasOut, weightsOut); // TODO: AUTOMATE szPadded

        execTimeI += vonMisesStats.elapsedTimeMs();
        ROFL_VAR1(vonMisesStats.elapsedTimeMs());
    }
    rofl::Profiler::getProfiler().printStats(std::cout);

    /********************START OF CUDA-RELATED PART************************/
    // CUDA MALLOC! -> vmm
    int mukwSz = musVmm.size(); //= szPadded
    ROFL_VAR1(mukwSz);          // !! MANDATORY PADDING!

    ROFL_ASSERT(mukwSz == szPadded && mukwSz == kappasVmm.size() && mukwSz == weightsVmm.size())

    int totalVmmSz = musVmm.size() * 2 * nSamples * nSamples; // musOut.size() * 2
    double *vmmDevice;
    hipMalloc((void **)&vmmDevice, totalVmmSz * sizeof(double));
    ROFL_VAR1(totalVmmSz); // !! MANDATORY PADDING!
    // hipMemcpy(kernelInput, dataChunk.data(), (dataChunk.size()) * sizeof (cuars::Vec2d), hipMemcpyHostToDevice);
    // hipMemset()

    // return 0;

    // CUDA MALLOC! -> mus
    double3 *musDevice;
    hipMalloc((void **)&musDevice, mukwSz * sizeof(double3));
    hipMemcpy(musDevice, musVmm.data(), (musVmm.size()) * sizeof(double3), hipMemcpyHostToDevice); // hipMemcpy on double3 ??
    // CUDA MALLOC! -> k
    double *kDevice;
    hipMalloc((void **)&kDevice, mukwSz * sizeof(double));
    hipMemcpy(kDevice, kappasVmm.data(), (kappasVmm.size()) * sizeof(double), hipMemcpyHostToDevice);

    // kernel input params:
    //  double *vmm, int nSamples, //!! mu size needs to be 3 times the size of k, w
    //  const double3 *mu, const double *k, const double *w, const int mukwSz
    int numBlocks = szPadded * 2 * nSamples * nSamples / numThreads; // TODO: automate numBlocks computation
    ROFL_VAR1(numBlocks);
    hipEvent_t startKernelAll, stopKernelAll; // timing using CUDA events
    hipEventCreate(&startKernelAll);
    hipEventCreate(&stopKernelAll);
    hipEventRecord(startKernelAll);
    vmm3d_kernel<<<numBlocks, numThreads>>>(vmmDevice, nSamples, musDevice, kDevice, szPadded);
    hipEventRecord(stopKernelAll);

    // std::vector<double> vmmHost(totalVmmSz, 0.0);
    // hipMemcpy(vmmHost.data(), vmmDevice, (totalVmmSz) * sizeof(double), hipMemcpyDeviceToHost);

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

    hipEventSynchronize(stopKernelAll);
    float millisecondsKernelAll = 0.0f;
    hipEventElapsedTime(&millisecondsKernelAll, startKernelAll, stopKernelAll);
    std::cout << "millisecondsKernelAll " << millisecondsKernelAll << " ms" << std::endl;

    hipEventDestroy(startKernelAll);
    hipEventDestroy(stopKernelAll);

    hipFree(kDevice);
    hipFree(musDevice);

    // maxVal = vmmHost.empty() ? -1 : *std::max_element(vmmHost.begin(), vmmHost.end());
    // minVal = vmmHost.empty() ? -1 : *std::min_element(vmmHost.begin(), vmmHost.end());
    // int maxId = vmmHost.empty()
    //                 ? -1
    //                 : std::distance(vmmHost.begin(),
    //                                 std::max_element(vmmHost.begin(), vmmHost.end()));

    // ROFL_VAR3(maxId, minVal, maxVal);

    // SUMMATION
    // CUDA MALLOC! -> w
    double *wDevice;
    hipMalloc((void **)&wDevice, mukwSz * sizeof(double));
    // cuars::VecVec2d dataChunk(points.begin() + indicesStartEnd.first, points.begin() + (indicesStartEnd.first + currChunkSz));
    hipMemcpy(wDevice, weightsVmm.data(), (weightsVmm.size()) * sizeof(double), hipMemcpyHostToDevice);
    // CUDA MALLOC! -> vmmSums
    double *vmmSumsDevice;
    hipMalloc((void **)&vmmSumsDevice, nSamples * 2 * nSamples * sizeof(double));
    hipMemset(vmmSumsDevice, 0.0, nSamples * 2 * nSamples);

    hipEvent_t startKernelDevice, stopKernelDevice; // timing using CUDA events
    hipEventCreate(&startKernelDevice);
    hipEventCreate(&stopKernelDevice);
    hipEventRecord(startKernelDevice);
    hipEventRecord(stopKernelDevice);
    vmm3d_summation_kernel<<<2 * nSamples * nSamples, 1>>>(vmmSumsDevice, nSamples, //!! mu size needs to be 3 times the size of k, w
                                                           vmmDevice, wDevice, mukwSz);

    std::vector<double> vmm(nSamples * 2 * nSamples, 0.0);
    hipMemcpy(vmm.data(), vmmSumsDevice, (nSamples * 2 * nSamples) * sizeof(double), hipMemcpyDeviceToHost);

    cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

    hipEventSynchronize(stopKernelDevice);
    float millisecondsKernelDevice = 0.0f;
    hipEventElapsedTime(&millisecondsKernelDevice, startKernelDevice, stopKernelDevice);
    std::cout << "millisecondsKernelDevice " << millisecondsKernelDevice << " ms" << std::endl;

    hipEventDestroy(startKernelDevice);
    hipEventDestroy(stopKernelDevice);

    // int i = 0;
    // for (auto &v : vmm)
    // {
    //     ROFL_VAR2(i, v)
    //     ++i;
    // }

    hipFree(wDevice);
    hipFree(vmmDevice);
    /********************END OF CUDA-RELATED PART************************/

    // for (size_t i = 0; i < vmm.size(); ++i)
    // {
    // ROFL_VAR3(i, 360.0 * i / nSamples, vmm[i]);
    // }

    /**
     * Plot VMM distribution values
     */
    // dsd::plotVmm(vsmm, minVal, maxVal, viewer);

    double maxVal = vmm.empty() ? -1 : *std::max_element(vmm.begin(), vmm.end());
    double minVal = vmm.empty() ? -1 : *std::min_element(vmm.begin(), vmm.end());
    int maxId = vmm.empty()
                    ? -1
                    : std::distance(vmm.begin(),
                                    std::max_element(vmm.begin(), vmm.end()));
    ROFL_VAR3(maxId, minVal, maxVal)

    /**
     * Find peaks of VMM
     */
    ROFL_VAR1("von Mises max")
    std::vector<int> vmmMaxima;
    std::vector<double> maximaValues;
    std::vector<std::pair<double, double>> thetaPhiMaxima;
    {
        rofl::ScopedTimer vonMisesMax("von Mises max");
        dsd::fvmMax(vmmMaxima, maximaValues, thetaPhiMaxima, vmm, nSamples, angleWin);
        execTimeI += vonMisesMax.elapsedTimeMs();
        ROFL_VAR1(vonMisesMax.elapsedTimeMs())
    }

    int maxIdx = 0;
    for (auto &tpm : thetaPhiMaxima)
    {
        // ROFL_VAR3(maximaValues[maxIdx], tpm.first, tpm.second)
        maxIdx++;
    }

    // plot von Mises mixture vector(matrix)
    double dtheta = M_PI / nSamples;

    int sizePhis = 2 * nSamples;
    int sizeThetas = nSamples;
    Eigen::MatrixXd vmmEigenMat(sizePhis, sizeThetas);
    for (int jTheta = 0; jTheta < sizeThetas; ++jTheta)
    {
        // double theta = dtheta * jTheta;

        for (int jPhi = 0; jPhi < sizePhis; ++jPhi)
        {
            // int j = jTheta * sizePhis + jPhi;
            // double phi = dtheta * jPhi;
            vmmEigenMat(jPhi, jTheta) = vmm[jTheta * sizePhis + jPhi];
        }
    }
    std::ofstream fileOutVmm("vmm.csv");
    fileOutVmm << vmmEigenMat << std::endl;
    fileOutVmm.close();

    if (enablePclVisualization)
    {
        pcl::visualization::PCLVisualizer::Ptr viewer(
            new pcl::visualization::PCLVisualizer("3D Viewer"));
        viewer->setBackgroundColor(0.9, 0.9, 0.9);
        viewer->addCoordinateSystem(2.0);
        viewer->initCameraParameters();

        viewer->addPointCloud<pcl::PointXYZ>(cloud, "cloud");

        viewer->setPointCloudRenderingProperties(
            pcl::visualization::PCL_VISUALIZER_POINT_SIZE, 1, "cloud");
        viewer->setPointCloudRenderingProperties(
            pcl::visualization::PCL_VISUALIZER_COLOR, 0.05, 0.05, 0.05, "cloud");

        // pass to viewer

        // viewer->removeAllPointClouds();
        // viewer->removeAllShapes();

        std::cout << "Plotting result of gme" << std::endl;

        // Plot Ellipses for GMM covariance matrices and relative max eigenvector
        // direction on PCL visualizer
        if (plotEllipsesArrows)
            dsd::plotEllipsesArrows3d(viewer, musOut, sigmasOut, weightsOut);

        while (!viewer->wasStopped())
        {
            viewer->spinOnce(100);
            // ROFL_VAR1("viewer->spinOnce(100)")
        }
    }

    return 0;
}